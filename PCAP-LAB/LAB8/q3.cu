#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void countWordFreq(char *str, char *word, int *freq, int len_word){
    int space = 0, start_index, end_index, len_word_device;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	char *ptr_start = str, *ptr_end, *ptr_space;
	ptr_space = ptr_start;
	while((*ptr_space) != ' ' && (*ptr_space) != '\0'){
		ptr_space++;
	}
	space++;

	while(space <= id){
		ptr_start = ptr_space + 1;
    ptr_space = ptr_start;
		while((*ptr_space) != ' ' && (*ptr_space) != '\0'){
      ptr_space++;
    }
		space++;
	}

	ptr_end = ptr_space - 1;
  	start_index = ptr_start - str;
	end_index = ptr_end - str;
	len_word_device = end_index-start_index+1;
    
    if(len_word_device == len_word){
        int count = len_word;
        int i = 0;
        while(count != 0){
            if(str[start_index+i] != word[i]){
                break;
            }
            else{
                i++;
                count--;
            }
        }
        if(count == 0){
            atomicAdd(freq,1);
        }
    }
}

int main(){
    char str[1000] = "Quick A Quick Brown Fox Quick Quick Jumps Over The Lazy dog Quick";
    char word[1000] = "Quick";
    int num_words = 13;

    /*
	printf("enter number of words\n");
	scanf("%d",&num_words);

	printf("enter string\n");
	scanf("%s",str);

    printf("enter word\n");
	scanf("%s",word);
	*/

    int size_char = sizeof(char);
    int size_int = sizeof(int);
    int len_str = strlen(str);
    int len_word = strlen(word);

    int freq=0;

    char *d_a, *d_b;
    int *d_c;
    hipMalloc((void **)&d_a,size_char*(strlen(str)+1));
    hipMalloc((void**)&d_b,size_char*(strlen(word)+1));
    hipMalloc((void**)&d_c,size_int*1);

    hipMemcpy(d_a,str,size_char*(strlen(str)+1),hipMemcpyHostToDevice);
    hipMemcpy(d_b,word,size_char*(strlen(word)+1),hipMemcpyHostToDevice);

    countWordFreq<<<num_words,1>>>(d_a,d_b,d_c,len_word);

    hipMemcpy(&freq,d_c,size_int*1,hipMemcpyDeviceToHost);

    printf("frequency of '%s' in '%s' is %d\n",word,str,freq);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;

}