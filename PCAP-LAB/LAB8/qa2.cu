#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
__global__ void repeatEachCharInxexNumberOfTimes(char *inputString, char *outputString)
{
	int index = threadIdx.x;
	int startIndex = (index*(index+1))/2;
	int numberOfTimes = index+1;
	char toBeRepeated = inputString[index];

	for (int i = 0; i < numberOfTimes; ++i)
	{
		outputString[startIndex + i] = toBeRepeated;
	}
}

int main(void)
{
	char inputString[100], outputString[100];
	printf("Enter the string: "); gets(inputString);
	int inputStringLength = strlen(inputString);
	int outputStringLength = ((inputStringLength*(inputStringLength+1)/2)*sizeof(char));

	char *d_inputString, *d_outputString;
	// int *d_inputStringLength;

	hipMalloc((void**)&d_inputString, inputStringLength*sizeof(char));
	// hipMalloc((void**)&d_inputStringLength,1*sizeof(int));
	hipMalloc((void**)&d_outputString, outputStringLength*sizeof(char));

	hipMemcpy(d_inputString, inputString, inputStringLength*sizeof(char), hipMemcpyHostToDevice);
	// hipMemcpy(d_inputStringLength, &inputStringLength, 1*sizeof(int), hipMemcpyHostToDevice);

	repeatEachCharInxexNumberOfTimes<<<1,inputStringLength>>>(d_inputString, d_outputString);

	hipMemcpy(outputString, d_outputString, outputStringLength*sizeof(char), hipMemcpyDeviceToHost);
	outputString[outputStringLength] = '\0';
	printf("Output: %s\n", outputString);
	

	hipFree(d_inputString);
	// hipFree(d_inputStringLength);
	hipFree(d_outputString);
	return 0;
}