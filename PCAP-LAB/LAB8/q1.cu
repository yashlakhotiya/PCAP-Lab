#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>

__global__ void stringCopy(char *S, char  *B, int N){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = id*N;
	for(int i=0; i<N; i++){
		B[offset + i] = S[i];
	}
}

int main(){
	char S[1000] = "hello",B[1000];
	int N = 3;
	/*printf("enter string\n");
	scanf("%s",S);
	printf("enter N\n");
	scanf("%d",&N);*/
	char *d_a, *d_b;

	int size = sizeof(char);

	hipMalloc((void **)&d_a,size * strlen(S));
	hipMalloc((void **)&d_b,size * N * strlen(S));

	hipMemcpy(d_a,S, size*strlen(S), hipMemcpyHostToDevice);

	stringCopy<<<N,1>>>(d_a,d_b,strlen(S));

	hipMemcpy(B,d_b,size*strlen(S)*N,hipMemcpyDeviceToHost);

	printf("str S: %s\n",S);
	printf("str B: %s\n",B);
	printf("\n");

	hipFree(d_a);
	hipFree(d_b);
	return 0;
}