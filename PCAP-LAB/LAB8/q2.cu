#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void reverseWord(char *str, char *rev_str){
	int space = 0, start_index, end_index, len_word;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	char *ptr_start = str, *ptr_end, *ptr_space;
	ptr_space = ptr_start;
	while((*ptr_space) != ' ' && (*ptr_space) != '\0'){
		ptr_space++;
	}
	space++;

	while(space <= id){
		ptr_start = ptr_space + 1;
    ptr_space = ptr_start;
		while((*ptr_space) != ' ' && (*ptr_space) != '\0'){
      ptr_space++;
    }
		space++;
	}

	ptr_end = ptr_space - 1;
  	start_index = ptr_start - str;
	end_index = ptr_end - str;
	len_word = end_index-start_index+1;

	for(int i=start_index,j=end_index; i<=end_index,j>=start_index; i++,j--){
		rev_str[i] = str[j];
	}
	if(id == gridDim.x-1){
		rev_str[end_index+1] = '\0';
	}
	else{
		rev_str[end_index+1] = ' ';
	}
	printf("id: %d, st_ind: %d, end_ind: %d, len: %d\n",id,start_index,end_index,len_word);

}
//string should end with a space
int main(){
	char str[1000] = "A Quick Brown Fox Jumps Over The Lazy dog",	rev_str[1000];
	int num_words = 9;
	printf("%s\n",str);
	for(int i=0;i<strlen(str); i++){
		printf("%d",i%10);
	}
	printf("\n");
	/*
	printf("enter number of words\n");
	scanf("%d",&num_words);

	printf("enter string\n");
	scanf("%s",str);
	*/
  	int size = sizeof(char);
	int len = strlen(str)+1;

	char *d_a, *d_b;

	hipMalloc((void **)&d_a,size*len);
	hipMalloc((void **)&d_b,size*len);

	hipMemcpy(d_a,str,size*len,hipMemcpyHostToDevice);

	reverseWord<<<num_words,1>>>(d_a,d_b);

	hipMemcpy(rev_str,d_b,size*len,hipMemcpyDeviceToHost);

	printf("reversed string is: %s\n",rev_str);

	hipFree(d_a);
	hipFree(d_b);
}