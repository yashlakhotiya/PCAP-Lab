#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
__global__ void repeatString(char *inputString, int inputStringLength, char *outputString, int n)
{
	int i = threadIdx.x;

	for(int count = 0, j = i ; count < n ; count++, j+=inputStringLength)
	{
		outputString[j] = inputString[i];
	}
}

int main(void)
{
	char inputString[100], outputString[100];
	printf("Enter the string: "); gets(inputString);
	int inputStringLength = strlen(inputString);

	int n; printf("Enter N: "); scanf("%d",&n);

	char *d_inputString, *d_outputString;

	hipMalloc((void**)&d_inputString, inputStringLength*sizeof(char));
	hipMalloc((void**)&d_outputString,n*inputStringLength*sizeof(char));

	hipMemcpy(d_inputString, inputString, inputStringLength*sizeof(char), hipMemcpyHostToDevice);

	repeatString<<<1,inputStringLength>>>(d_inputString, inputStringLength, d_outputString, n);

	hipMemcpy(outputString, d_outputString, n*inputStringLength*sizeof(char), hipMemcpyDeviceToHost);
	outputString[n*inputStringLength] = '\0';
	printf("Output: %s\n", outputString);
	

	hipFree(d_inputString);
	hipFree(d_outputString);
	return 0;
}