#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>

__global__ void matrixop(int *a, int *t)
{
	int n= threadIdx.x,m=blockIdx.x, size=blockDim.x,size1=gridDim.x;
	t[m*size+n]=powf(a[m*size+n],m+1);
}

int main(void)
{
	int *a,*t,m,n,i,j;
	int *d_a,*d_t;
	printf("Enter the value of m:\n");scanf("%d",&m);
	printf("Enter the value of n:\n");scanf("%d",&n);

	int size=sizeof(int)*m*n;
	a=(int*)malloc(m*n*sizeof(int));
	t=(int*)malloc(m*n*sizeof(int));

	printf("Enter input matrix:\n");
	for(i=0;i<m*n;i++)
		scanf("%d",&a[i]);
	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_t,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

	matrixop<<<m,n>>>(d_a,d_t);
	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result vector is:\n");
	for(i=0;i<m;i++)
	{
		for(j=0;j<n;j++)
			printf("%d\t",t[i*n+j]);
		printf("\n");
	}
	getchar();
	hipFree(d_a);
	hipFree(d_t);
	return 0;


}