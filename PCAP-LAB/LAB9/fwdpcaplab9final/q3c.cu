
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void MatMul(int *a, int *b, int *t, int m0, int n0, int m1, int n1)
{
	int nthColumn = threadIdx.x, mthRow = blockIdx.x;

	int temp = 0;
	for( int i = 0; i < n0 ; i++ )
	{
		temp += a[mthRow*n0 + i] * b[nthColumn + i*n1];
		// printf("%d %d: %d %d\n", mthRow, nthColumn, a[mthRow*n0 + i], b[nthColumn + i*n1]);
	}
	t[mthRow*n1 + nthColumn] = temp;

	


}
int main() {
	int *a, *b, *t, m0, n0, m1, n1, i, j;
	int *d_a, *d_b, *d_t;

	printf("Enter value of m0\n"); scanf("%d", &m0);
	printf("Enter value of n0\n"); scanf("%d", &n0);
	int size0 = sizeof(int)*m0*n0;

	printf("Enter value of m1\n"); scanf("%d", &m1);
	printf("Enter value of n1\n"); scanf("%d", &n1);
	int size1 = sizeof(int)*m1*n1;

	int sizet = sizeof(int)*m0*n1;

	if(n0!=m1)
	{
		printf("Invalid matrix dimensions.\n");
		exit(0);
	}

	a = (int *)malloc(m0*n0*sizeof(int));
	b = (int *)malloc(m1*n1*sizeof(int));
	t = (int *)malloc(m0*n1*sizeof(int));


	printf("Enter input matrix A\n");
	for(i=0; i< m0*n0; i++)
		scanf("%d", &a[i]);

	printf("Enter input matrix B\n");
	for(i=0; i< m1*n1; i++)
		scanf("%d", &b[i]);


	hipMalloc((void**)&d_a, size0);
	hipMalloc((void**)&d_b, size1);
	hipMalloc((void**)&d_t, sizet);

	hipMemcpy(d_a, a, size0, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size1, hipMemcpyHostToDevice);

	MatMul<<<m0,n1>>>(d_a, d_b, d_t, m0, n0, m1, n1);

	hipMemcpy(t, d_t, sizet, hipMemcpyDeviceToHost);

	printf("result vector:\n");
	for(i=0; i<m0; i++) {
		for(j =0; j<n1; j++)
			printf("%d ", t[i*n1+j]);
		printf("\n");
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_t);
	return 0;



}

// 2 3 3 2 1 2 3 4 5 6 1 2 3 4 5 6