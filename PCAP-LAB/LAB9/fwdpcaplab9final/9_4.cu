#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void func4(int* a, int* t)
{
	int n = threadIdx.x;
	int m = blockIdx.x;
	int size = blockDim.x;
	int size1 = gridDim.x;

	int dx = a[m*size+n];
	if ((n != 0) && (m != 0) && (n != size-1) && (m != size1-1))
	{
		int gx;
		int ax = 0;
		int fac = 1;
		while (dx != 0)
		{
			gx = dx%2;
			if (gx == 0)
			{
				ax += fac;
			}
			fac *= 10;
			dx /= 2;
		}

		t[m*size+n] = ax;
	}
	else
	{
		t[m*size+n] = dx;
	}
}

int main(void)
{
	int *a,*t,m,n,i,j;
	int *d_a,*d_t;

	printf("Enter value of m: ");
	scanf("%d",&m);
	printf("Enter value of n: ");
	scanf("%d",&n);

	int size = sizeof(int)*m*n;
	a = (int*)malloc(m*n*sizeof(int));
	t = (int*)malloc(m*n*sizeof(int));

	printf("Enter input matrix:\n");
	for (i = 0; i < m*n; i++)
	{
		scanf("%d",&a[i]);
	}

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_t,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	func4<<<m,n>>>(d_a,d_t);
	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);

	printf("The result vector is:\n");
	for (i = 0; i < m; i++)
	{
		for (j = 0; j < n; j++)
		{
			printf("%d\t",t[i*n+j]);
		}
		printf("\n");
	}

	getchar();
	hipFree(d_a);
	hipFree(d_t);
	return 0;
}