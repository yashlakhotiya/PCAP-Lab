#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>

__global__ void addmat(int *a, int *b, int *c)
{
	int n= threadIdx.x,m=blockIdx.x, size=blockDim.x;
	c[m*size+n]=a[m*size+n]+b[m*size+n];
}
__global__ void addrow (int *A, int *B, int *C,int n) {
    int idx = threadIdx.x;
    printf("idx = %d\n", idx);
    for (int i = 0; i < n; ++i) {
        C[i + n * idx] = A[i + n * idx] + B[i + n * idx];
    }
}
__global__ void addcol(int *A, int *B, int *C,int m) {
    int idx = threadIdx.x;
    int x=blockDim.x;
    printf("idx = %d\n", idx);
    for (int i = 0; i < m; ++i) {
        C[ i*x + idx] = A[ i*x + idx] + B[ i*x + idx];
				
    }
}



int main(void)
{
	int a[8]={1,2,3,4,5,6,1,2},b[8]={1,2,3,4,5,6,1,2},*c,*c1,*c2,m=4,n=2,i,j;
	int *d_a,*d_b,*d_c,*d_c1,*d_c2;
	

	int size=sizeof(int)*m*n;
	
	c=(int*)malloc(m*n*sizeof(int));
	c1=(int*)malloc(m*n*sizeof(int));
	c2=(int*)malloc(m*n*sizeof(int));


	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMalloc((void**)&d_c,size);
	hipMalloc((void**)&d_c1,size);
	hipMalloc((void**)&d_c2,size);
	
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
  addrow<<<1, m>>>(d_a, d_b, d_c,n);	
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

	printf("Result matrix using computation using each row is:\n");
	for(i=0;i<m;i++)
	{
		for(j=0;j<n;j++)
			printf("%d\t",c[i*n+j]);
		printf("\n");
	}


	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	addcol<<<1,n>>>(d_a,d_b,d_c2,m);
	hipMemcpy(c2,d_c2,size,hipMemcpyDeviceToHost);
	
	
	printf("Result matrix using computation using each column is:\n");
	for(i=0;i<m;i++)
	{
		for(j=0;j<n;j++)
			printf("%d\t",c2[i*n+j]);
		printf("\n");
	}


	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	addmat<<<m,n>>>(d_a,d_b,d_c1);
	hipMemcpy(c1,d_c1,size,hipMemcpyDeviceToHost);
	
	
	printf("Result matrix using computation using each element is:\n");
	for(i=0;i<m;i++)
	{
		for(j=0;j<n;j++)
			printf("%d\t",c1[i*n+j]);
		printf("\n");
	}
	getchar();
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_c1);
	return 0;


}
