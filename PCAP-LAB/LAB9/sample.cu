#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>


__global__ void transpose(int *a, int *t){
    int n = threadIdx.x;
    int m = blockIdx.x;
    int size = blockDim.x;
    int size1 = gridDim.x;
    t[n*size1+m] = a[m*size+n];
}

int main(){
    int m=3,n=2;
    int a[m*n] = {1,2,3,4,5,6};
    int t[m*n];
    
    /*
    int *a,*t,m=3,n=2;
    printf("enter the value of m and n: \n");
    scanf("%d%d",&m,&n);
    a = (int *)malloc(m*n*sizeof(int));
    c = (int *)malloc(m*n*sizeof(int));
    printf("enter input matrix\n");
    for(int i=0; i<m*n; i++){
        scanf("%d",&a[i]);
    }
    */

    int *d_a, *d_t;
    int size = sizeof(int)*m*n;

    hipMalloc((void **)&d_a,size);
    hipMalloc((void **)&d_t,size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    transpose<<<m,n>>>(d_a,d_t);
    hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
    printf("result vector is:\n");
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            printf("%d\t",t[i*m+j]);
        }
        printf("\n");
    }
    hipFree(d_a);
    hipFree(d_t);
    return 0;
}