#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>


__global__ void row_sum(int *a, int *b, int *res){
    int i = blockIdx.x;
    int n = blockDim.x;//number of columns
    for(int k=0; k<n; k++){
        res[i*n+k] = a[i*n+k] + b[i*n+k];
    }
}

__global__ void column_sum(int *a, int *b, int *res){
    int j = threadIdx.x;
    int m = gridDim.x;//number of rows
    for(int k=0; k<m; k++){
        res[j*m+k] = a[j*m+k] + b[j*m+k];
    }
}

__global__ void cell_sum(int *a, int *b, int *res){
    int i = blockIdx.x;
    int j = threadIdx.x;
    int m = gridDim.x;
    int n = blockDim.x;

    res[i*n+j] = a[i*n+j] + b[i*n+j];
}

int main(){
    int m=3,n=2;
    int a[m*n] = {1,2,3,4,5,6};
    int b[m*n] = {1,2,3,4,5,6};
    int res[m*n];
    
    /*
    int *a,*b,*res,m=3,n=2;
    printf("enter the value of m and n: \n");
    scanf("%d%d",&m,&n);
    a = (int *)malloc(m*n*sizeof(int));
    c = (int *)malloc(m*n*sizeof(int));
    printf("enter input matrix\n");
    for(int i=0; i<m*n; i++){
        scanf("%d",&a[i]);
    }
    */

    int *d_a, *d_b, *d_res;
    int size = sizeof(int)*m*n;

    hipMalloc((void **)&d_a,size);
    hipMalloc((void **)&d_b,size);
    hipMalloc((void **)&d_res,size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
    row_sum<<<m,n>>>(d_a,d_b,d_res);
    hipMemcpy(res,d_res,size,hipMemcpyDeviceToHost);
    printf("row sum : result vector is:\n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%d\t",res[i*n+j]);
        }
        printf("\n");
    }

    column_sum<<<m,n>>>(d_a,d_b,d_res);
    hipMemcpy(res,d_res,size,hipMemcpyDeviceToHost);
    printf("column sum : result vector is:\n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%d\t",res[i*n+j]);
        }
        printf("\n");
    }

    cell_sum<<<m,n>>>(d_a,d_b,d_res);
    hipMemcpy(res,d_res,size,hipMemcpyDeviceToHost);
    printf("cell sum : result vector is:\n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%d\t",res[i*n+j]);
        }
        printf("\n");
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);
    return 0;
}