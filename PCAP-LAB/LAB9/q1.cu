#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>


__global__ void row_power(int *a, int *t){
    int n = threadIdx.x;
    int m = blockIdx.x;
    int size = blockDim.x;//number of columns
    int size1 = gridDim.x;//number of rows
    t[m*size+n] = pow(a[m*size+n],m+1);
}

int main(){
    int m=3,n=2;
    int a[m*n] = {1,2,3,4,5,6};
    int t[m*n];
    
    /*
    int *a,*t,m=3,n=2;
    printf("enter the value of m and n: \n");
    scanf("%d%d",&m,&n);
    a = (int *)malloc(m*n*sizeof(int));
    c = (int *)malloc(m*n*sizeof(int));
    printf("enter input matrix\n");
    for(int i=0; i<m*n; i++){
        scanf("%d",&a[i]);
    }
    */

    int *d_a, *d_t;
    int size = sizeof(int)*m*n;

    hipMalloc((void **)&d_a,size);
    hipMalloc((void **)&d_t,size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    row_power<<<m,n>>>(d_a,d_t);
    hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
    printf("result vector is:\n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%d\t",t[i*n+j]);
        }
        printf("\n");
    }
    hipFree(d_a);
    hipFree(d_t);
    return 0;
}